/* CUDA FFT Library */
/* written by Viktor K. Decyk, UCLA */

#include <stdlib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>

extern int nblock_size;
extern int maxgsx;

static hipError_t crc;
static hipfftResult cfrc;
static hipfftHandle planrx, planxr, planrxn, planxrn;
static hipfftHandle plany, planyn;

__global__ void gpuctpose4(float2 f[], float2 g[], int nx, int ny,
                           int nxv, int nyv);

__global__ void gpuctpose4n(float2 fn[], float2 gn[], int nx, int ny,
                            int ndim, int nxv, int nyv);

/*--------------------------------------------------------------------*/
__global__ void gpusctpose4(float2 f[], float2 g[], float ani, int nx,
                            int ny, int nxv, int nyv) {
/* scaled complex transpose using blocking algorithm with gaps */
/* local data */
   int j, k, js, ks, joff, koff, mx, mxv;
   float2 a;
/* The size of the shared memory array is as follows: */
/* float2 shm2[(mx + 1)*mx];                          */
   extern __shared__ float2 shm2[];
   mx = blockDim.x;
   mxv = mx + 1;
   joff = mx*blockIdx.x;
   koff = mx*blockIdx.y;
   js = threadIdx.x;
   ks = threadIdx.y;
/* copy into block */
   j = js + joff;
   k = ks + koff;
   if ((j < nx) && (k < ny)) {
      shm2[js+mxv*ks] = f[j+nxv*k];
   }
   __syncthreads();
/* copy out from block with scaling */
   j = ks + joff;
   k = js + koff;
   if ((j < nx) && (k < ny)) {
      a = shm2[ks+mxv*js];
      a.x = ani*a.x;
      a.y = ani*a.y;
      g[k+nyv*j] = a;
   }
   return;
}

/*--------------------------------------------------------------------*/
__global__ void gpusctpose4n(float2 fn[], float2 gn[], float ani,
                             int nx, int ny, int ndim, int nxv,
                             int nyv) {
/* scaled complex vector transpose using blocking algorithm with gaps */
/* ndim = vector dimension                                            */
/* local data */
   int i, j, k, js, ks, joff, koff, mx, mxv, nmxv, nnxv, nnyv, jj, kk;
   float2 a;
/* The size of the shared memory array is as follows: */
/* float2 shmn2[ndim*(mx + 1)*mx];                    */
   extern __shared__ float2 shmn2[];
   mx = blockDim.x;
   mxv = mx + 1;
   joff = mx*blockIdx.x;
   koff = mx*blockIdx.y;
   js = threadIdx.x;
   ks = threadIdx.y;
   nmxv = ndim*mxv;
   nnxv = ndim*nxv;
   nnyv = ndim*nyv;
/* copy into block */
   j = js + joff;
   k = ks + koff;
   if ((j < nx) && (k < ny)) {
      jj = j + nnxv*k;
      kk = js + nmxv*ks;
      for (i = 0; i < ndim; i++) {
         shmn2[kk+mxv*i] = fn[jj+nxv*i];
      }
   }
   __syncthreads();
/* copy out from block with scaling */
   j = ks + joff;
   k = js + koff;
   if ((j < nx) && (k < ny)) {
      kk = k + nnyv*j;
      jj = ks + nmxv*js;
      for (i = 0; i < ndim; i++) {
         a = shmn2[jj+mxv*i];
         a.x = ani*a.x;
         a.y = ani*a.y;
         gn[kk+nyv*i] = a;
      }
   }
   return;
}

/*--------------------------------------------------------------------*/	
extern "C" void gpufft2rrcuinit(int nx, int ny, int ndim) {
   cfrc = hipfftPlan1d(&planrx,nx,HIPFFT_R2C,ny);
   if (cfrc) {
      printf("hipfftPlan1d planrx error=%d\n",cfrc);
      exit(1);
   }
   cfrc = hipfftPlan1d(&planxr,nx,HIPFFT_C2R,ny);
   if (cfrc) {
      printf("hipfftPlan1d planxr error=%d\n",cfrc);
      exit(1);
   }
   cfrc = hipfftPlan1d(&planrxn,nx,HIPFFT_R2C,ndim*ny);
   if (cfrc) {
      printf("hipfftPlan1d planrxn error=%d\n",cfrc);
      exit(1);
   }
   cfrc = hipfftPlan1d(&planxrn,nx,HIPFFT_C2R,ndim*ny);
   if (cfrc) {
      printf("hipfftPlan1d planxrn error=%d\n",cfrc);
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/	
extern "C" void gpufft2cuinit(int nx, int ny, int ndim) {
   int nxh1;
   nxh1 = nx/2 + 1;
   cfrc = hipfftPlan1d(&plany,ny,HIPFFT_C2C,nxh1);
   if (cfrc) {
      printf("hipfftPlan1d plany error=%d\n",cfrc);
      exit(1);
   }
   cfrc = hipfftPlan1d(&planyn,ny,HIPFFT_C2C,ndim*nxh1);
   if (cfrc) {
      printf("hipfftPlan1d planyn error=%d\n",cfrc);
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpufft2rrcudel() {
   cfrc = hipfftDestroy(planrx);
   if (cfrc) {
      printf("hipfftDestroy planrx error=%d\n",cfrc);
      exit(1);
   }
   cfrc = hipfftDestroy(planxr);
   if (cfrc) {
      printf("hipfftDestroy planxr error=%d\n",cfrc);
      exit(1);
   }
   cfrc = hipfftDestroy(planrxn);
   if (cfrc) {
      printf("hipfftDestroy planrxn error=%d\n",cfrc);
      exit(1);
   }
   cfrc = hipfftDestroy(planxrn);
   if (cfrc) {
      printf("hipfftDestroy planxrn error=%d\n",cfrc);
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpufft2cudel() {
   cfrc = hipfftDestroy(plany);
   if (cfrc) {
      printf("hipfftDestroy plany error=%d\n",cfrc);
      exit(1);
   }
   cfrc = hipfftDestroy(planyn);
   if (cfrc) {
      printf("hipfftDestroy planyn error=%d\n",cfrc);
      exit(1);
   }
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpufft2rrcu(float2 f[], float2 g[], int isign,
                            int indx, int indy, int nxh1d, int nyd) {
/* wrapper function for real to complex fft, without packed data */
/* uses 1D real to complex and complex to complex NVIDIA FFTs    */
/* nxh1d must be = nx/2+1                                        */
/* local data */
   int nx, nxh1, ny, ns;
   int mx = 16;
   float ani;
   dim3 dimBlock(nblock_size);
   dim3 dimBlockt(mx,mx);
/* calculate range of indices */
   nx = 1L<<indx;
   nxh1 = nx/2 + 1;
   ny = 1L<<indy;
   dim3 dimGridtx((nxh1-1)/mx+1,(ny-1)/mx+1);
   dim3 dimGridty((ny-1)/mx+1,(nxh1-1)/mx+1);
   ns = (mx+1)*mx*sizeof(float2);
/* inverse fourier transform */
   if (isign < 0) {
/* perform x fft */
      cfrc = hipfftExecR2C(planrx,(hipfftReal *)f,(hipfftComplex *)f);
/*    cudaDeviceSynchronize(); */
      if (cfrc) {
         printf("cufftExecR2C(-1) planrx error=%d\n",cfrc);
         exit(1);
      }
/* transpose f to g and normalize */
      ani = 1.0f/(((float) nx)*((float) ny));
      crc = hipGetLastError();
      gpusctpose4<<<dimGridtx,dimBlockt,ns>>>(f,g,ani,nxh1,ny,nxh1d,
                                              nyd);
/*    cudaDeviceSynchronize(); */
      crc = hipGetLastError();
      if (crc) {
         printf("gpusctpose4 error=%d:%s\n",crc,
                hipGetErrorString(crc));
         exit(1);
      }
/* perform y fft */
      cfrc = hipfftExecC2C(plany,(hipfftComplex *)g,(hipfftComplex *)g,
                          HIPFFT_FORWARD);
      hipDeviceSynchronize();
      if (cfrc) {
         printf("cufftExecC2C(-1) plany error=%d\n",cfrc);
         exit(1);
      }
   }
/* forward fourier transform */
   else if (isign > 0) {
/* perform y fft */
      cfrc = hipfftExecC2C(plany,(hipfftComplex *)g,(hipfftComplex *)g,
                          HIPFFT_BACKWARD);
/*    cudaDeviceSynchronize(); */
      if (cfrc) {
         printf("cufftExecC2C(1) plany error=%d\n",cfrc);
         exit(1);
      }
/* transpose g to f */
      crc = hipGetLastError();
      gpuctpose4<<<dimGridty,dimBlockt,ns>>>(g,f,ny,nxh1,nyd,nxh1d);
/*    cudaDeviceSynchronize(); */
      crc = hipGetLastError();
      if (crc) {
         printf("gpuctpose4 error=%d:%s\n",crc,hipGetErrorString(crc));
         exit(1);
      }
/* perform x fft */
      cfrc = hipfftExecC2R(planxr,(hipfftComplex *)f,(hipfftReal *)f);
      hipDeviceSynchronize();
      if (cfrc) {
         printf("cufftExecC2R(1) planxr error=%d\n",cfrc);
         exit(1);
      }
   }
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpufft2rrcun(float2 fn[], float2 gn[], int isign,
                             int indx, int indy, int ndim, int nxh1d,
                             int nyd) {
/* wrapper function for real to complex fft, without packed data */
/* for vector data                                               */
/* uses 1D real to complex and complex to complex NVIDIA FFTs    */
/* ndim = vector dimension                                       */
/* nxh1d must be = nx/2+1                                        */
/* local data */
   int nx, nxh1, ny, ns;
   int mx = 16;
   float ani;
   dim3 dimBlock(nblock_size);
   dim3 dimBlockt(mx,mx);
/* calculate range of indices */
   nx = 1L<<indx;
   nxh1 = nx/2 + 1;
   ny = 1L<<indy;
   dim3 dimGridtx((nxh1-1)/mx+1,(ny-1)/mx+1);
   dim3 dimGridty((ny-1)/mx+1,(nxh1-1)/mx+1);
   ns = ndim*(mx+1)*mx*sizeof(float2);
/* inverse fourier transform */
   if (isign < 0) {
/* perform x fft */
      cfrc = hipfftExecR2C(planrxn,(hipfftReal *)fn,(hipfftComplex *)fn);
/*    cudaDeviceSynchronize(); */
      if (cfrc) {
         printf("cufftExecR2C(-1) planrxn error=%d\n",cfrc);
         exit(1);
      }
/* transpose f to g and normalize */
      ani = 1.0f/(((float) nx)*((float) ny));
      crc = hipGetLastError();
      gpusctpose4n<<<dimGridtx,dimBlockt,ns>>>(fn,gn,ani,nxh1,ny,ndim,
                                               nxh1d,nyd);
/*    cudaDeviceSynchronize(); */
      crc = hipGetLastError();
      if (crc) {
         printf("gpusctpose4n error=%d:%s\n",crc,
                hipGetErrorString(crc));
         exit(1);
      }
/* perform y fft */
      cfrc = hipfftExecC2C(planyn,(hipfftComplex *)gn,(hipfftComplex *)gn,
                          HIPFFT_FORWARD);
      hipDeviceSynchronize();
      if (cfrc) {
         printf("cufftExecC2C(-1) planyn error=%d\n",cfrc);
         exit(1);
      }
   }
/* forward fourier transform */
   else if (isign > 0) {
/* perform y fft */
      cfrc = hipfftExecC2C(planyn,(hipfftComplex *)gn,(hipfftComplex *)gn,
                          HIPFFT_BACKWARD);
/*    cudaDeviceSynchronize(); */
      if (cfrc) {
         printf("cufftExecC2C(1) planyn error=%d\n",cfrc);
         exit(1);
      }
/* transpose g to f */
      crc = hipGetLastError();
      gpuctpose4n<<<dimGridty,dimBlockt,ns>>>(gn,fn,ny,nxh1,ndim,nyd,
                                              nxh1d);
/*    cudaDeviceSynchronize(); */
      crc = hipGetLastError();
      if (crc) {
         printf("gpuctpose4n error=%d:%s\n",crc,
                hipGetErrorString(crc));
         exit(1);
      }
/* perform x fft */
      cfrc = hipfftExecC2R(planxrn,(hipfftComplex *)fn,(hipfftReal *)fn);
      hipDeviceSynchronize();
      if (cfrc) {
         printf("cufftExecC2R(1) planxrn error=%d\n",cfrc);
         exit(1);
      }
   }
   return;
}

/* Interfaces to Fortran */

/*--------------------------------------------------------------------*/	
extern "C" void gpufft2rrcuinit_(int *nx, int *ny, int *ndim) {
   gpufft2rrcuinit(*nx,*ny,*ndim);
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpufft2cuinit_(int *nx, int *ny, int *ndim) {
   gpufft2cuinit(*nx,*ny,*ndim);
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpufft2rrcudel_() {
   gpufft2rrcudel();
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpufft2cudel_() {
   gpufft2cudel();
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpufft2rrcu_(unsigned long *gp_f, unsigned long *gp_g,
                             int *isign, int *indx, int *indy,
                             int *nxh1d, int *nyd) {
   float2 *f, *g;
   f = (float2 *)*gp_f;
   g = (float2 *)*gp_g;
   gpufft2rrcu(f,g,*isign,*indx,*indy,*nxh1d,*nyd);
   return;
}

/*--------------------------------------------------------------------*/
extern "C" void gpufft2rrcun_(unsigned long *gp_fn,
                              unsigned long *gp_gn, int *isign,
                              int *indx, int *indy, int *ndim,
                              int *nxh1d, int *nyd) {
   float2 *fn, *gn;
   fn = (float2 *)*gp_fn;
   gn = (float2 *)*gp_gn;
   gpufft2rrcun(fn,gn,*isign,*indx,*indy,*ndim,*nxh1d,*nyd);
   return;
}

